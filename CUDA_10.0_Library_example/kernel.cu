#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "example.cuh"

#include <stdio.h>

namespace simple_cuda_example {

	hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

	__global__ void addKernel(int *c, const int *a, const int *b)
	{
		int i = threadIdx.x;
		c[i] = a[i] + b[i];
	}
	
	__device__ void sayHello(int i) {
		printf("[EXAMPLE]: Hi, I am device function called from the %d thread!\n", threadIdx.x);
		if (i == threadIdx.x) {
			printf(" - index = thread id: (%d = %d)\n",i,threadIdx.x);
		} else {
			printf(" - index != thread id: (%d = %d); are you using Grid-Stride Loop?\n", i, threadIdx.x);
		}
	}

	int example()
	{
		const int arraySize = 5;
		const int a[arraySize] = { 1, 2, 3, 4, 5 };
		const int b[arraySize] = { 10, 20, 30, 40, 50 };
		int c[arraySize] = { 0 };

		// Add vectors in parallel.
		hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "[EXAMPLE]: addWithCuda failed!");
			return 1;
		}

		printf("[EXAMPLE]: {1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
			c[0], c[1], c[2], c[3], c[4]);

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "[EXAMPLE]: hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}

	// Helper function for using CUDA to add vectors in parallel.
	hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
	{
		int *dev_a = 0;
		int *dev_b = 0;
		int *dev_c = 0;
		hipError_t cudaStatus;

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "[EXAMPLE]: hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "[EXAMPLE]: hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "[EXAMPLE]: hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "[EXAMPLE]: hipMalloc failed!");
			goto Error;
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "[EXAMPLE]: hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "[EXAMPLE]: hipMemcpy failed!");
			goto Error;
		}

		// Launch a kernel on the GPU with one thread for each element.
		addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "[EXAMPLE]: addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "[EXAMPLE]: hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "[EXAMPLE]: hipMemcpy failed!");
			goto Error;
		}

	Error:
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);

		return cudaStatus;
	}

}
